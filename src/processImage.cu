#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <ctime>
#include <vector>
#include <stdio.h>
#include "CImg.h"

#include "processImage.h"
#include "Filters.h"
#include "Utils.h"
#include "Constants.h"
using namespace cimg_library;

template<typename T>
__device__ T DomainCheck(T pixel)
{
  PIXEL_DOMAIN_CHECK(pixel);
  return pixel;
}

template <typename T>
__global__ void processingKernel(T* inputBuffer, T* outputBuffer, int width, 
                            int height, int channels,const int offset)
{
  
  int redChannelOffset = blockIdx.x * blockDim.x + threadIdx.x;
  int greenChannelOffset = redChannelOffset + 1*offset;
  int blueChannelOffset = redChannelOffset + 2*offset;
  float brightnessVal = 0.0f;
  outputBuffer[redChannelOffset] = (T)DomainCheck<float>(50+inputBuffer[redChannelOffset]);
  outputBuffer[greenChannelOffset] = (T)DomainCheck<float>(50+inputBuffer[greenChannelOffset]);
  outputBuffer[blueChannelOffset] = (T)DomainCheck<float>(50+inputBuffer[blueChannelOffset]);
  __syncthreads();
}

struct Setup
{
  int threads;
  int blocks;
};

unsigned int powerOf2( unsigned int x ) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}


void getSetupConfig(unsigned int problemSize, struct Setup* setup)
{
  int threads = powerOf2(problemSize);
  threads = threads <= MAX_THREADS ? threads : MAX_THREADS;
  
  int blocks = (problemSize)/threads;
  setup->threads = threads;
  setup->blocks = blocks;
}

void sendWarmUpSignal(unsigned char* h_data, const unsigned int sizeData)
{
  unsigned char* d_data;
  hipMalloc((void**)&d_data,sizeData);
  hipMemcpy(d_data,h_data,sizeData,hipMemcpyHostToDevice);
  hipFree(d_data);
}

void runKernel(unsigned char* h_data, unsigned char* h_result,const unsigned int width, const unsigned int height, unsigned int channels)
{
  int problemSize = width*height*channels;
  int sizeData = problemSize*sizeof(unsigned char);
  int sizeResult = sizeData;
  
  Setup setup;
  getSetupConfig(width*height,&setup);
  std::cout << "Blocks: " << setup.blocks << std::endl;
  std::cout << "Threads: " << setup.threads << std::endl;

  unsigned char *d_data;
  hipMalloc((void**)&d_data,sizeData);
  hipMemcpy(d_data,h_data,sizeData,hipMemcpyHostToDevice);
  
  unsigned char* d_result;
  hipMalloc((void**)&d_result,sizeData);
  
  dim3 dimGrid(setup.blocks,1,1);
  dim3 dimBlock(setup.threads,1,1);
  int sizeSharedMem = problemSize;
  int offset = width*height;
  processingKernel<unsigned char><<<dimGrid,dimBlock>>>(d_data,d_result,width,height,channels, offset);
  
  hipMemcpy(h_result,d_result,sizeResult,
             hipMemcpyDeviceToHost);
  
  hipFree(d_data);
  hipFree(d_result);
}


void printMetaData(const gpu::Image& image)
{
  std::cout << "Image Metadata:" << std::endl;
  std::cout << "width: " << image.width << ", height: " << image.height << 
  ", size: " << image.size << ", spectrum: " << image.spectrum << std::endl;
}

int main(int argc, char* argv[])
{
  //cimg::imagemagick_path("/opt/local/bin/convert");
  if(argc == 3)
  {
    std::string filename = argv[1];
    std::string outputFilename = argv[2];
    CImg<unsigned char> image(filename.c_str());
    CImgDisplay mainDisplay(image,"Image",0);

    gpu::Image imgInfo(image.width(),image.height(),image.width()*image.height(),image.spectrum());
    printMetaData(imgInfo);
    
    /*
     <summary> 
     1. Allocate Buffers
     2. Get Meta information from the image and assign that to ImageInfo object.
     3. Copy image into Input Buffer (unroll operation).
     4. Perform the operation.
     */
    
    unsigned char* inputBuffer = new unsigned char[imgInfo.spectrum*imgInfo.size];
    unsigned char* outputBuffer = new unsigned char[imgInfo.spectrum*imgInfo.size];
    
    gpu::ImageProcessing<unsigned char> imp;
    timeval tim;
    
    double dTime1 = gpu::getTime(tim);
    
    gpu::unroll(image,imgInfo.width,imgInfo.height,imgInfo.spectrum,
                inputBuffer);
    
    sendWarmUpSignal(inputBuffer,imgInfo.width*imgInfo.height*imgInfo.spectrum); 
    double dTime2 = gpu::getTime(tim);
    std::cout << "time taken for unrolled version: " << dTime2 - dTime1 << std::endl;
    runKernel(inputBuffer,outputBuffer,imgInfo.width, imgInfo.height, imgInfo.spectrum);
    
    //imp.saturation(S_VALUE,inputBuffer, outputBuffer, 
      //             imgInfo.width, imgInfo.height, imgInfo.spectrum);
    
    CImg<unsigned char> outputImage(outputBuffer,imgInfo.width,imgInfo.height,1,
                                    imgInfo.spectrum,0);
    
    double dTime3 = gpu::getTime(tim);
    std::cout << "time taken for unrolled version: " << dTime3 - dTime2 << std::endl;
    
    outputImage.save_jpeg(outputFilename.c_str());
    CImgDisplay darkDisplay(outputImage,"Output Image",0);
    
    while(!(mainDisplay.is_closed()))
    {
      mainDisplay.wait();
    }
    delete[] inputBuffer;
    delete[] outputBuffer;
  }
  else
  {
    std::cout << "Usage: " << argv[0] << " <image-filename> <output-filename>" << std::endl;
  }
}
